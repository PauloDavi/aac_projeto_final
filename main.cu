
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <hiprand/hiprand.h>

using namespace std;

#define NUM_THREAD 512
#define N_RANDOMS 1000000
#define N_CITIES 26

std::string cities[N_CITIES] = {
  "joao_pessoa",
  "aracaju",
  "belem",
  "belo_horizonte",
  "boa_vista",
  "brasília",
  "campo_grande",
  "cuiaba",
  "curitiba",
  "florianopolis",
  "fortaleza",
  "goiania",
  "maceio",
  "manaus",
  "natal",
  "palmas",
  "porto_alegre",
  "porto_velho",
  "recife",
  "rio_branco",
  "rio_de_janeiro",
  "salvador",
  "sao_luis",
  "sao_paulo",
  "teresina",
  "vitoria"};

__device__ int distances[N_CITIES][N_CITIES] = {
  {0, 611, 2161, 2171, 6593, 2245, 3357, 3366, 3188, 3485, 688, 2442, 395, 5808, 185, 2253, 3889, 4822, 120, 5356, 2448, 949, 1660, 2770, 1224, 2001},
  {611, 0, 2079, 1578, 6000, 1652, 2765, 2775, 2595, 2892, 1183, 1848, 294, 5215, 788, 1662, 3296, 4230, 501, 4763, 1855, 356, 1578, 2187, 1142, 1408},
  {2161, 2079, 0, 2824, 6083, 2120, 2942, 2941, 3193, 3500, 1610, 2017, 2173, 5298, 2108, 1283, 3852, 4397, 2074, 4931, 3250, 2100, 806, 2933, 947, 3108},
  {2171, 1578, 2824, 0, 4736, 716, 1453, 1594, 1004, 1301, 2528, 906, 1854, 3951, 2348, 1690, 1712, 3050, 2061, 3584, 434, 1372, 2738, 586, 2302, 524},
  {6593, 6000, 6083, 4736, 0, 4275, 3836, 3142, 4821, 5128, 6548, 4076, 6279, 785, 6770, 4926, 5348, 1686, 6483, 2230, 5159, 5794, 6120, 4756, 6052, 5261},
  {2245, 1652, 2120, 716, 4275, 0, 1134, 1133, 1366, 1673, 2200, 209, 1930, 3490, 2422, 973, 2027, 2589, 2135, 3123, 1148, 1446, 2157, 1015, 1789, 1239},
  {3357, 2765, 2942, 1453, 3836, 1134, 0, 694, 991, 1298, 3407, 935, 3040, 3051, 3534, 1785, 1518, 2150, 3247, 2684, 1444, 2568, 2979, 1014, 2911, 1892},
  {3366, 2775, 2941, 1594, 3142, 1133, 694, 0, 1679, 1986, 3406, 934, 3049, 2357, 3543, 1784, 2206, 1456, 3255, 1990, 2017, 2566, 2978, 1614, 2910, 2119},
  {3188, 2595, 3193, 1004, 4821, 1366, 991, 1679, 0, 300, 3541, 1186, 2871, 4036, 3365, 2036, 711, 3135, 3078, 3669, 852, 2385, 3230, 408, 3143, 1300},
  {3485, 2892, 3500, 1301, 5128, 1673, 1298, 1986, 300, 0, 3838, 1493, 3168, 4443, 3662, 2336, 476, 3442, 3375, 3976, 1144, 2682, 3537, 705, 3450, 1597},
  {688, 1183, 1610, 2528, 6548, 2200, 3407, 3406, 3541, 3838, 0, 2482, 1075, 5763, 537, 2035, 4242, 4862, 800, 5396, 2805, 1389, 1070, 3127, 634, 2397},
  {2442, 1848, 2017, 906, 4076, 209, 935, 934, 1186, 1493, 2482, 0, 2125, 3291, 2618, 874, 1847, 2390, 2332, 2924, 1338, 1643, 2054, 926, 1986, 1428},
  {395, 294, 2173, 1854, 6279, 1930, 3040, 3049, 2871, 3168, 1075, 2125, 0, 5491, 572, 1851, 3572, 4505, 285, 5039, 2131, 632, 1672, 2453, 1236, 1684},
  {5808, 5215, 5298, 3951, 785, 3490, 3051, 2357, 4036, 4443, 5763, 3291, 5491, 0, 5985, 4141, 4563, 901, 5698, 1445, 4374, 5009, 5335, 3971, 5267, 4476},
  {185, 788, 2108, 2348, 6770, 2422, 3534, 3543, 3365, 3662, 537, 2618, 572, 5985, 0, 2345, 4066, 4998, 297, 5533, 2625, 1126, 1607, 2947, 1171, 2178},
  {2253, 1662, 1283, 1690, 4926, 973, 1785, 1784, 2036, 2336, 2035, 874, 1851, 4141, 2345, 0, 2747, 0, 2058, 3764, 2124, 1454, 1386, 1776, 1401, 2214},
  {3889, 3296, 3852, 1712, 5348, 2027, 1518, 2206, 711, 476, 4242, 1847, 3572, 4563, 4066, 2747, 0, 3662, 3779, 4196, 1553, 3090, 3891, 1109, 3804, 2001},
  {4822, 4230, 4397, 3050, 1686, 2589, 2150, 1456, 3135, 3442, 4862, 2390, 4505, 901, 4998, 0, 3662, 0, 4712, 544, 3473, 4023, 4434, 3070, 4366, 3575},
  {120, 501, 2074, 2061, 6483, 2135, 3247, 3255, 3078, 3375, 800, 2332, 285, 5698, 297, 2058, 3779, 4712, 0, 5243, 2338, 839, 1573, 2660, 1137, 1831},
  {5356, 4763, 4931, 3584, 2230, 3123, 2684, 1990, 3669, 3976, 5396, 2924, 5039, 1445, 5533, 3764, 4196, 544, 5243, 0, 4007, 4457, 4968, 3604, 4900, 4109},
  {2448, 1855, 3250, 434, 5159, 1148, 1444, 2017, 852, 1144, 2805, 1338, 2131, 4374, 2625, 2124, 1553, 3473, 2338, 4007, 0, 1649, 3015, 429, 2579, 521},
  {949, 356, 2100, 1372, 5794, 1446, 2568, 2566, 2385, 2682, 1389, 1643, 632, 5009, 1126, 1454, 3090, 4023, 839, 4457, 1649, 0, 1599, 1962, 1163, 1202},
  {1660, 1578, 806, 2738, 6120, 2157, 2979, 2978, 3230, 3537, 1070, 2054, 1672, 5335, 1607, 1386, 3891, 4434, 1573, 4968, 3015, 1599, 0, 2970, 446, 2607},
  {2770, 2187, 2933, 586, 4756, 1015, 1014, 1614, 408, 705, 3127, 926, 2453, 3971, 2947, 1776, 1109, 3070, 2660, 3604, 429, 1962, 2970, 0, 2792, 882},
  {1224, 1142, 947, 2302, 6052, 1789, 2911, 2910, 3143, 3450, 634, 1986, 1236, 5267, 1171, 1401, 3804, 4366, 1137, 4900, 2579, 1163, 446, 2792, 0, 2171},
  {2001, 1408, 3108, 524, 5261, 1239, 1892, 2119, 1300, 1597, 2397, 1428, 1684, 4476, 2178, 2214, 2001, 3575, 1831, 4109, 521, 1202, 2607, 882, 2171, 0}};

__device__ uint8_t rand_num(float *randoms, int index, int min = 1, int max = N_CITIES)
{
  index = index % N_RANDOMS;
  float random = randoms[index];
  return (uint8_t)(min + random * (max - min));
}

__device__ int calc_path_length(uint8_t *stops)
{
  int length = 0;
  for (int i = 0; i < N_CITIES; i++)
    length += distances[stops[i]][stops[i + 1]];
  return length;
}

string stops_to_path(uint8_t *stops)
{
  string path;
  for (int i = 0; i <= N_CITIES; i++)
  {
    path += cities[stops[i]];
    if (i < N_CITIES)
      path += " -> ";
  }
  return path;
}

__global__ void init_population(uint8_t *population, int *path_size, float *randoms, int pop_size)
{
  // Sequential thread index across the blocks
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= pop_size)
    return;
  auto index_stop = idx * (N_CITIES + 1);
  auto stops = population + index_stop;
  stops[0] = 0;
  for (int i = 1; i < N_CITIES; i++)
    stops[i] = i;
  for (int i = 1; i < N_CITIES - 1; i++) {
    auto n = rand_num(randoms, index_stop + i, i);
    if (n != i) {
      auto temp = stops[i];
      stops[i] = stops[n];
      stops[n] = temp;
    }
  }
  stops[N_CITIES] = 0;
  path_size[idx] = calc_path_length(stops);
}

__global__ void tsp_calc(uint8_t *population, int *path_size, int gen_thres, float *randoms, int pop_size)
{
  // Sequential thread index across the blocks
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= pop_size)
    return;
  auto stops = population + idx * (N_CITIES + 1);
  for (int gen = 1; gen <= gen_thres; gen++)
  {
    int rand_1 = rand_num(randoms, idx + gen * 2);
    int rand_2 = rand_num(randoms, idx + gen * 2 + 1);
    if (rand_1 == rand_2)
      continue;
    auto temp = stops[rand_1];
    stops[rand_1] = stops[rand_2];
    stops[rand_2] = temp;
    auto new_path_size = calc_path_length(stops);
    if (new_path_size <= path_size[idx])
      path_size[idx] = new_path_size;
    else
    {
      stops[rand_2] = stops[rand_1];
      stops[rand_1] = temp;
    }
  }
}

__global__ void minReduce(int *path_size, int pop_size, int *result_index)
{
  int ti = threadIdx.x;
  __shared__ volatile float min_value;
  __shared__ volatile float min_index;
  if (ti == 0) min_value = 999999;
  for (int i = ti; i < pop_size; i += NUM_THREAD)
  {
    if (i >= pop_size) break;
    float v = path_size[i];
    __syncthreads();
    while (v < min_value) {
      min_value = v;
      min_index = i;
    }
    __syncthreads();
  }
  if (ti == 0) result_index[0] = min_index;
}

// Main routine that executes on the host
int main(int argc, char *argv[])
{
  if (argc != 3)
  {
    cout << "Invalid args";
    return -1;
  }
  int gen_thres = atoi(argv[1]);
  int pop_size = atoi(argv[2]);

  uint8_t *population;
  int *path_size_gpu;
  float *randoms;
  auto mem_path_size = pop_size * sizeof(int);
  hipMalloc((void **)&population, pop_size * (N_CITIES + 1));
  hipMalloc((void **)&path_size_gpu, mem_path_size);
  hipMalloc((void **)&randoms, N_RANDOMS * sizeof(float));

  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));
  hiprandGenerateUniform(gen, randoms, N_RANDOMS);

  auto num_blocks = ceil(float(pop_size) / NUM_THREAD);
  init_population<<<num_blocks, NUM_THREAD>>>(population, path_size_gpu, randoms, pop_size);
  tsp_calc<<<num_blocks, NUM_THREAD>>>(population, path_size_gpu, gen_thres, randoms, pop_size);

  int *best_index_gpu;
  int best_index = 0;
  hipMalloc((void **)&best_index_gpu, sizeof(int));
  minReduce<<<1, NUM_THREAD>>>(path_size_gpu, pop_size, best_index_gpu);
  hipMemcpy(&best_index, best_index_gpu, sizeof(int), hipMemcpyDeviceToHost);

  int best_path_size = 0;
  uint8_t *best = (uint8_t *)malloc(N_CITIES + 1);
  hipMemcpy(&best_path_size, path_size_gpu + best_index, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(best, population + best_index * (N_CITIES + 1), N_CITIES + 1, hipMemcpyDeviceToHost);

  cout << "Best solution (" << best_path_size << " km"
       << "): "
       << stops_to_path(best) << endl;

  // Cleanup
  free(best);
  hipFree(best_index_gpu);
  hipFree(population);
  hipFree(path_size_gpu);
  hiprandDestroyGenerator(gen);
  hipFree(randoms);

  return 0;
}